#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#define STB_IMAGE_WRITE_IMPLEMENTATION
#define __STDC_LIB_EXT1__
#include "stb_image_write.h"

#include <iostream>
#include <ctime>
#include <vector>

#include "rtweekend.h"
#include "hittable.h"
#include "hittable_list.h"
#include "sphere.h"
#include <new>

__device__ color ray_color(const ray& r, hittable_list **world) {
  hit_record rec;
  if ((*world)->hit(r, { 0.f, infinity }, rec)) {
    return 0.5f * (rec.normal + color(1.f, 1.f, 1.f));
  }

  vec3 unit_direction = unit_vector(r.direction());
  auto a = 0.5f * (unit_direction.y() + 1.0f);
  return (1.0f - a) * color(1.0, 1.0, 1.0) + a * color(0.5, 0.7, 1.0);
}

__global__ void render(vec3 *fb, int width, int height, vec3 to_pixel00, vec3 viewport_u, vec3 viewport_v, vec3 camera_center,
                       hittable_list **world) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if (i >= width || j >= height) return;
  int pixel_index = (j * width + i);

  //instead of pixel_delta, divide everytime for better accuracy
  auto ray_direction = to_pixel00 + (i * viewport_u)/width + (j * viewport_v)/height; 
  ray r(camera_center, ray_direction);
  fb[pixel_index] = ray_color(r, world);
}

__global__ void create_world(hittable **d_list, hittable_list **d_world) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *(d_list) = new sphere(vec3(0, 0, -1), 0.5);
    *(d_list + 1) = new sphere(vec3(0, -100.5, -1), 100);
    *d_world = new hittable_list(d_list, 2);
  }
}

__global__ void free_world(hittable **d_list, hittable_list **d_world) {
  delete *(d_list);
  delete *(d_list + 1);
  delete *d_world;
}

int main() {
  auto a = sizeof(interval);
  // Image
  auto aspect_ratio = 16.0 / 9.0;
  int nx = 1200;
  int tx = 8;
  int ty = 8;

  // Calculate the image height, and ensure that it's at least 1.
  int ny = int(nx / aspect_ratio);
  ny = (ny < 1) ? 1 : ny;

  // Camera

  auto focal_length = 1.0;
  auto viewport_height = 2.0;
  auto viewport_width = viewport_height * (double(nx) / ny);
  auto camera_center = point3(0, 0, 0);

  // Calculate the vectors across the horizontal and down the vertical viewport edges.
  auto viewport_u = vec3(viewport_width, 0, 0);
  auto viewport_v = vec3(0, -viewport_height, 0);

  // Calculate the horizontal and vertical delta vectors from pixel to pixel.
  auto pixel_delta_u = viewport_u / nx;
  auto pixel_delta_v = viewport_v / ny;

  // Calculate the location of the upper left pixel.
  auto viewport_upper_left = camera_center
    - vec3(0, 0, focal_length) - viewport_u / 2 - viewport_v / 2;
  auto pixel00_loc = viewport_upper_left + 0.5 * (pixel_delta_u + pixel_delta_v);
  auto to_pixel00 = pixel00_loc - camera_center;

  // Render
  std::cout << "Rendering a " << nx << "x" << ny << " image ";
  std::cout << "in " << tx << "x" << ty << " blocks.\n";

  int num_pixels = nx * ny;
  std::vector<uint8_t> imageBuffer; imageBuffer.reserve(num_pixels * 3);
  size_t fb_size = num_pixels * sizeof(vec3);

  // allocate FB
  vec3 *fb;
  checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

  // make our world of hitables
  hittable **d_list;
  checkCudaErrors(hipMalloc((void **)&d_list, 2 * sizeof(hittable *)));
  hittable_list **d_world;
  checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable_list *)));
  create_world << <1, 1 >> > (d_list, d_world);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  clock_t start, stop;
  start = clock();
  // Render our buffer
  dim3 blocks(nx / tx + 1, ny / ty + 1);
  dim3 threads(tx, ty);
  render << <blocks, threads >> > (fb, nx, ny, to_pixel00, viewport_u, viewport_v, camera_center, d_world);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  stop = clock();
  double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
  std::cout << "took " << timer_seconds << " seconds.\n";

  for (int j = 0; j < ny; ++j) {
    for (int i = 0; i < nx; ++i) {
      int pixel_index = (j * nx + i);
      write_color(imageBuffer, fb[pixel_index]);
    }
  }

  if (stbi_write_bmp("rendered.bmp", nx, ny, 3, imageBuffer.data()) == 1) {
    std::cout << "image out success\n";
  }
  else {
    std::cout << "image out fail\n";
  }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}
